#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

__host__ __device__ int outInvariant(int inValue) {
  return inValue * inValue;
}

__host__ __device__ int outDependent(int value, int inIdx, int outIdx) {
  if (inIdx == outIdx) {
    return 2 * value;
  } else if (inIdx > outIdx) {
    return value / (inIdx - outIdx);
  } else {
    return value / (outIdx - inIdx);
  }
}

__global__ void s2g_gpu_gather_kernel(int *in, int *out, int len) {
  //@@ INSERT CODE HERE
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int output = 0;
  if(i<len){
    for(int j = 0; j< len; j++){
      int intermediate = outInvariant(in[j]);
      output += outDependent(intermediate,j,i);
    }
    out[i] = output;
  }
}

static void s2g_cpu_scatter(int *in, int *out, int len) {
  for (int inIdx = 0; inIdx < len; ++inIdx) {
    int intermediate = outInvariant(in[inIdx]);
    for (int outIdx = 0; outIdx < len; ++outIdx) {
      out[outIdx] += outDependent(intermediate, inIdx, outIdx);
    }
  }
}

static void s2g_gpu_gather(int *in, int *out, int len) {
  //@@ INSERT CODE HERE
  dim3 dimblock(1024,1,1);
  dim3 dimgrid(ceil(len/1024.0),1,1);
  s2g_gpu_gather_kernel<<<dimgrid, dimblock>>>(in, out, len);
}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  int *hostInput;
  int *hostOutput;
  int *deviceInput;
  int *deviceOutput;
  size_t byteCount;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput = (int *)wbImport(wbArg_getInputFile(args, 0), &inputLength,
                              "Integer");
  hostOutput = (int *)malloc(inputLength * sizeof(int));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);
  byteCount = inputLength * sizeof(int);

  wbTime_start(GPU, "Allocating GPU memory.");
  wbCheck(hipMalloc((void **)&deviceInput, byteCount));
  wbCheck(hipMalloc((void **)&deviceOutput, byteCount));
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  wbCheck(hipMemcpy(deviceInput, hostInput, byteCount,
                     hipMemcpyHostToDevice));
  wbCheck(hipMemset(deviceOutput, 0, byteCount));
  wbTime_stop(GPU, "Copying input memory to the GPU.");


  //////////////////////////////////////////
  // GPU Gather Computation
  //////////////////////////////////////////
  wbTime_start(Compute, "Performing GPU Gather computation");
  s2g_gpu_gather(deviceInput, deviceOutput, inputLength);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing GPU Gather computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  wbCheck(hipMemcpy(hostOutput, deviceOutput, byteCount,
                     hipMemcpyDeviceToHost));
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbSolution(args, hostOutput, inputLength);

  wbTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceInput);
  hipFree(deviceOutput);
  wbTime_stop(GPU, "Freeing GPU Memory");

  free(hostInput);
  free(hostOutput);

  return 0;
}

