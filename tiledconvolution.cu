#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "CUDA error: ", hipGetErrorString(err));              \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      return -1;                                                          \
    }                                                                     \
  } while (0)

//@@ Define any useful program-wide constants here
#define MASK_WIDTH 3
#define TILE_WIDTH 8
#define INPUT_WIDTH 10
  typedef struct Matrix{
  unsigned int width;
  unsigned int height;
  unsigned int pitch;
  float * elements;
  }*mcMatrix;
//@@ Define constant memory for device kernel here
__constant__ float Mc[MASK_WIDTH][MASK_WIDTH][MASK_WIDTH];

__global__ void conv3d(float *input, float *output, const int z_size,
                       const int y_size, const int x_size) {
  //@@ Insert kernel code here
  __shared__ float Ns[INPUT_WIDTH][INPUT_WIDTH][INPUT_WIDTH];
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  int row_o = blockIdx.y * TILE_WIDTH +ty;
  int col_o = blockIdx.x * TILE_WIDTH +tx;
  int z_o = blockIdx.z * TILE_WIDTH + tz;
  int row_i = row_o -1;
  int col_i = col_o -1;
  int z_i = z_o -1;
  float outcome = 0.0f;
  if((row_i>=0)&&(col_i<x_size)&&(col_i>=0)&&(row_i<y_size)&&(z_i>=0)&&(z_i<z_size)){
    Ns[tz][ty][tx] = input[z_i*(x_size*y_size)+row_i*x_size+col_i];
  }
  else 
    Ns[tz][ty][tx]=0.0f;
  
  __syncthreads();

if( ty<TILE_WIDTH && tx<TILE_WIDTH && tz<TILE_WIDTH){
  for(unsigned i=0;i<3;i++){
    for(unsigned j=0;j<3;j++){
      for(unsigned k=0;k<3;k++){
        outcome+=Mc[i][j][k]*Ns[i+tz][j+ty][k+tx];
      }
    }
  }

__syncthreads();
if(row_o<y_size&&col_o<x_size&&z_o<z_size){
  output[z_o*(x_size*y_size)+row_o*x_size+col_o] = outcome;
}
}
}


int main(int argc, char *argv[]) {
  wbArg_t args;
  int z_size;
  int y_size;
  int x_size;
  int inputLength, kernelLength;
  float *hostInput;
  float *hostKernel;
  float *hostOutput;
  float *deviceInput;
  float *deviceOutput;
 
  args = wbArg_read(argc, argv);

  // Import data
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostKernel =
      (float *)wbImport(wbArg_getInputFile(args, 1), &kernelLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  // First three elements are the input dimensions
  z_size = hostInput[0];
  y_size = hostInput[1];
  x_size = hostInput[2];
  wbLog(TRACE, "The input size is ", z_size, "x", y_size, "x", x_size);
  assert(z_size * y_size * x_size == inputLength - 3);
  assert(kernelLength == 27);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  //@@ Allocate GPU memory here
  // Recall that inputLength is 3 elements longer than the input data
  // because the first  three elements were the dimensions
  wbCheck(hipMalloc((void**)&deviceInput,(inputLength-3) * sizeof(float)));
  wbCheck(hipMalloc((void**)&deviceOutput,(inputLength-3) * sizeof(float)));
  
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  //@@ Copy input and kernel to GPU here
  // Recall that the first three elements of hostInput are dimensions and
  // do
  // not need to be copied to the gpu
   wbCheck(hipMemcpy(deviceInput,&hostInput[3],(inputLength-3) * sizeof(float),hipMemcpyHostToDevice));

   wbCheck(hipMemcpyToSymbol(HIP_SYMBOL(Mc),hostKernel,MASK_WIDTH*MASK_WIDTH*MASK_WIDTH*sizeof(float)));
  
  wbTime_stop(Copy, "Copying data to the GPU");

  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ Initialize grid and block dimensions here
  dim3 DimGrid(ceil(x_size/8.0), ceil(y_size/8.0), ceil(z_size/8.0));
  dim3 DimBlock(INPUT_WIDTH, INPUT_WIDTH, INPUT_WIDTH);
  
  //@@ Launch the GPU kernel here
  conv3d<<<DimGrid , DimBlock>>>(deviceInput , deviceOutput , z_size,y_size,x_size);
  
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Doing the computation on the GPU");

  wbTime_start(Copy, "Copying data from the GPU");
  //@@ Copy the device memory back to the host here
  // Recall that the first three elements of the output are the dimensions
  // and should not be set here (they are set below)
  wbCheck(hipMemcpy(&hostOutput[3],deviceOutput,(inputLength-3) * sizeof(float),hipMemcpyDeviceToHost));
  
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  // Set the output dimensions for correctness checking
  hostOutput[0] = z_size;
  hostOutput[1] = y_size;
  hostOutput[2] = x_size;
  wbSolution(args, hostOutput, inputLength);

  // Free device memory
  hipFree(deviceInput);
  hipFree(deviceOutput);

  // Free host memory
  free(hostInput);
  free(hostOutput);
  return 0;
}

