#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)

/// For simplicity, fix #bins=1024 so scan can use a single block and no
/// padding
#define NUM_BINS 1024
#define BLOCK_SIZE 512
/******************************************************************************
 GPU main computation kernels
*******************************************************************************/

__global__ void gpu_normal_kernel(float *in_val, float *in_pos, float *out,
                                  int grid_size, int num_in) {

  //@@ INSERT CODE HERE
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  float output = 0;
  if(i<grid_size){
    for(int j = 0; j < num_in; j++){
      const float dist = in_pos[j] - (float)i;
      const float in_val2 = in_val[j] * in_val[j];
      output = output + in_val2/(dist*dist);
    }
  out[i] = output;
  }
  
}

__global__ void gpu_cutoff_kernel(float *in_val, float *in_pos, float *out,
                                  int grid_size, int num_in,
                                  float cutoff2) {

  //@@ INSERT CODE HERE
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  float output = 0;
  if(i<grid_size){
    for(int j = 0; j < num_in; j++){
      const float dist = in_pos[j] - (float)i;
      if(dist > cutoff2) continue;
      const float in_val2 = in_val[j] * in_val[j];
      output = output + in_val2/(dist*dist);
    }
  out[i] = output;
  }
  
  
  
}

__global__ void gpu_cutoff_binned_kernel(int *bin_ptrs,
                                         float *in_val_sorted,
                                         float *in_pos_sorted, float *out,
                                         int grid_size, float cutoff2) {

  //@@ INSERT CODE HERE
  int i  = threadIdx.x + blockIdx.x*blockDim.x;
	
	float dist;
	float result=0;
	
	unsigned int iterator;
	float dist2;
	if(i<grid_size){
    for(int bin = 0; bin < NUM_BINS; ++bin) {
            unsigned int startOfBin = bin_ptrs[bin];
            unsigned int endOfBin = bin_ptrs[bin+1];
            dist = in_pos_sorted[startOfBin] - (float)i;
            dist2 = dist * dist;
            if(dist2 <= cutoff2) {
			/*checking for if the bin starting point is less than the cutoff or not. */
				for(iterator = startOfBin; iterator < endOfBin; ++iterator) {
					float point_dist = in_pos_sorted[iterator] - (float)i;
					float points_dist2 = point_dist * point_dist;
					if(points_dist2 <= cutoff2)
					result += (in_val_sorted[iterator] * in_val_sorted[iterator]) / points_dist2;
				}
	    }
	}
	out[i] = result;
		
	}
  
  
}

/******************************************************************************
 Main computation functions
*******************************************************************************/

void cpu_normal(float *in_val, float *in_pos, float *out, int grid_size,
                int num_in) {

  for (int inIdx = 0; inIdx < num_in; ++inIdx) {
    const float in_val2 = in_val[inIdx] * in_val[inIdx];
    for (int outIdx = 0; outIdx < grid_size; ++outIdx) {
      const float dist = in_pos[inIdx] - (float)outIdx;
      out[outIdx] += in_val2 / (dist * dist);
    }
  }
}

void gpu_normal(float *in_val, float *in_pos, float *out, int grid_size,
                int num_in) {

  const int numThreadsPerBlock = 512;
  const int numBlocks = (grid_size - 1) / numThreadsPerBlock + 1;
  gpu_normal_kernel<<<numBlocks, numThreadsPerBlock>>>(in_val, in_pos, out,
                                                       grid_size, num_in);
}

void gpu_cutoff(float *in_val, float *in_pos, float *out, int grid_size,
                int num_in, float cutoff2) {

  const int numThreadsPerBlock = 512;
  const int numBlocks = (grid_size - 1) / numThreadsPerBlock + 1;
  gpu_cutoff_kernel<<<numBlocks, numThreadsPerBlock>>>(
      in_val, in_pos, out, grid_size, num_in, cutoff2);
}

void gpu_cutoff_binned(int *bin_ptrs, float *in_val_sorted,
                       float *in_pos_sorted, float *out, int grid_size,
                       float cutoff2) {

  const int numThreadsPerBlock = 512;
  const int numBlocks = (grid_size - 1) / numThreadsPerBlock + 1;
  gpu_cutoff_binned_kernel<<<numBlocks, numThreadsPerBlock>>>(
      bin_ptrs, in_val_sorted, in_pos_sorted, out, grid_size, cutoff2);
}

/******************************************************************************
 Preprocessing kernels
*******************************************************************************/

__global__ void histogram(float *in_pos, int *bin_counts, int num_in,
                          int grid_size) {

  //@@ INSERT CODE HERE
/*   __shared__ unsigned int counts_private[1024];
  if(threadIdx.x<1024){
    histo_private[threadIdx.x] = 0;
  }
  __syncthreads();
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  while(i<num_in){
    const int binIdx = (int) ((in_pos[i]/grid_size)*NUM_BINS);
    atomicAdd(&(counts_private[binIdx]),1);
    i = i+stride;
  }
  __syncthreads();
  if(threadIdx.x<1024){
    atomicAdd(&(bin_counts[threadIdx.x]),counts_private[threadIdx.x]);
    
  }
  */
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  while(i<num_in){
    const int binIdx = (int) ((in_pos[i]/grid_size)*NUM_BINS);
    atomicAdd(&(bin_counts[binIdx]),1);
    i = i+stride;
  }
  
}

__global__ void scan(int *bin_counts, int *bin_ptrs) {

  //@@ INSERT CODE HERE
  
   __shared__ float T[BLOCK_SIZE*2];
 unsigned int t = threadIdx.x;
  unsigned int start = 2 * blockIdx.x * blockDim.x;
  
  int len = NUM_BINS;
  
  //load subarray into the shared subarray vector
  if((t+start)>=len)
    bin_counts[t+start]=0;
  if((start+BLOCK_SIZE+t)>=len)
    bin_counts[start+t+BLOCK_SIZE]=0;
  T[t]=bin_counts[t+start];
  T[t+BLOCK_SIZE]=bin_counts[start+BLOCK_SIZE+t];
  __syncthreads(); //wait for load to finish.
    
  int stride = 1;
  while(stride<=BLOCK_SIZE)
  {
    int index = (threadIdx.x+1)*stride*2-1;
    if(index<BLOCK_SIZE*2)
    { T[index] += T[index-stride];}
    stride = stride * 2;
    __syncthreads();
  }
  stride = BLOCK_SIZE/2;
  while(stride>0)
  {
    int index = (threadIdx.x+1)*stride*2-1;
    if((index+stride)<BLOCK_SIZE*2)
    {
      T[index+stride] += T[index];
    }
    stride = stride/2;
    __syncthreads();
  }
  
  if ((t+start) < len)
    bin_ptrs[t+start+1] = T[t];
  if ((start+BLOCK_SIZE+t) < len)
    bin_ptrs[start+BLOCK_SIZE+t+1] = T[t + BLOCK_SIZE];
}

__global__ void sort(float *in_val, float *in_pos, float *in_val_sorted,
                     float *in_pos_sorted, int grid_size, int num_in,
                     int *bin_counts, int *bin_ptrs) {

  //@@ INSERT CODE HERE
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int newIdx ;
  if(i<num_in){
    const int binIdx = (int)((in_pos[i] / grid_size) * NUM_BINS);
    newIdx = bin_ptrs[binIdx + 1] - atomicSub(&bin_counts[binIdx],1);
			in_val_sorted[newIdx] = in_val[i];
			in_pos_sorted[newIdx] = in_pos[i];
  }
   
}

/******************************************************************************
 Preprocessing functions
*******************************************************************************/

static void cpu_preprocess(float *in_val, float *in_pos,
                           float *in_val_sorted, float *in_pos_sorted,
                           int grid_size, int num_in, int *bin_counts,
                           int *bin_ptrs, float *in_pos_d, int *bin_counts_d, int *bin_ptrs_d) {

  // Histogram the input positions
 /* for (int inIdx = 0; inIdx < num_in; ++inIdx) {
    const int binIdx = (int)((in_pos[inIdx] / grid_size) * NUM_BINS);
    ++bin_counts[binIdx];  //counter, number of atmos in each bin
  }
*/
  //test histogram kernel
   const int numThreadsPerBlock = 512;
  histogram<<<30, numThreadsPerBlock>>>(in_pos_d, bin_counts_d, num_in,
                                        grid_size);
  hipMemcpy(bin_counts, bin_counts_d, NUM_BINS * sizeof(int),
                       hipMemcpyDeviceToHost);
  //end test
  //test scan 
   if (NUM_BINS != 1024) {
    wbLog(FATAL, "NUM_BINS must be 1024. Do not change.");
    return;
  }
  scan<<<1, numThreadsPerBlock>>>(bin_counts_d, bin_ptrs_d);
  hipMemcpy(bin_ptrs, bin_ptrs_d,
                         (NUM_BINS + 1) * sizeof(int),
                         hipMemcpyDeviceToHost);
  //end test scan
    
  
  /*
  // Scan the histogram to get the bin pointers
  bin_ptrs[0] = 0;
  for (int binIdx = 0; binIdx < NUM_BINS; ++binIdx) {
    bin_ptrs[binIdx + 1] = bin_ptrs[binIdx] + bin_counts[binIdx];
  }  //get the start indice
*/
  // Sort the inputs into the bins
  for (int inIdx = 0; inIdx < num_in; ++inIdx) {
    const int binIdx = (int)((in_pos[inIdx] / grid_size) * NUM_BINS);
    const int newIdx = bin_ptrs[binIdx + 1] - bin_counts[binIdx];   
    --bin_counts[binIdx];
    in_val_sorted[newIdx] = in_val[inIdx];
    in_pos_sorted[newIdx] = in_pos[inIdx];
  }
} //make a sorted new array 

static void gpu_preprocess(float *in_val, float *in_pos,
                           float *in_val_sorted, float *in_pos_sorted,
                           int grid_size, int num_in, int *bin_counts,
                           int *bin_ptrs) {

  const int numThreadsPerBlock = 512;

  // Histogram the input positions
  histogram<<<30, numThreadsPerBlock>>>(in_pos, bin_counts, num_in,
                                        grid_size);
  

  // Scan the histogram to get the bin pointers
  if (NUM_BINS != 1024) {
    wbLog(FATAL, "NUM_BINS must be 1024. Do not change.");
    return;
  }
  scan<<<1, numThreadsPerBlock>>>(bin_counts, bin_ptrs);

  // Sort the inputs into the bins
  sort<<<30, numThreadsPerBlock>>>(in_val, in_pos, in_val_sorted,
                                   in_pos_sorted, grid_size, num_in,
                                   bin_counts, bin_ptrs);
}

enum Mode {
  CPU_NORMAL = 1,
  GPU_NORMAL,
  GPU_CUTOFF,
  GPU_BINNED_CPU_PREPROCESSING,
  GPU_BINNED_GPU_PREPROCESSING
};

int main(int argc, char **argv) {
  wbArg_t args;

  // Initialize host variables
  // ----------------------------------------------

  // Variables
  float *in_val_h;
  float *in_pos_h;
  float *out_h;
  float *in_val_d;
  float *in_pos_d;
  float *out_d;
  int grid_size;
  int num_in;
  Mode mode;

  // Constants
  const float cutoff =
      3000.0f; // Cutoff distance for optimized computation
  const float cutoff2 = cutoff * cutoff;

  // Extras needed for input binning
  int *bin_counts_h;
  int *bin_ptrs_h;
  float *in_val_sorted_h;
  float *in_pos_sorted_h;
  int *bin_counts_d;
  int *bin_ptrs_d;
  float *in_val_sorted_d;
  float *in_pos_sorted_d;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  mode = (Mode)wbImport_flag(wbArg_getInputFile(args, 0));
  in_val_h = (float *)wbImport(wbArg_getInputFile(args, 1), &num_in);
  in_pos_h = (float *)wbImport(wbArg_getInputFile(args, 2), &num_in);
  grid_size = (int)wbImport_flag(wbArg_getInputFile(args, 3));

  out_h = (float *)calloc(grid_size, sizeof(float));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The number of inputs is ", num_in);
  wbLog(TRACE, "The grid_size is ", grid_size);
  
  //test
//  mode == GPU_BINNED_GPU_PREPROCESSING;

  // CPU Preprocessing
  // ------------------------------------------------------

  if (mode == GPU_BINNED_CPU_PREPROCESSING) {

    wbTime_start(Generic, "Allocating data for preprocessing");
    // Data structures needed to preprocess the bins on the CPU
    bin_counts_h = (int *)malloc(NUM_BINS * sizeof(int));
    bin_ptrs_h = (int *)malloc((NUM_BINS + 1) * sizeof(int));
    in_val_sorted_h = (float *)malloc(num_in * sizeof(float));
    in_pos_sorted_h = (float *)malloc(num_in * sizeof(float));

    //test, 
     wbCheck(hipMalloc((void **)&in_pos_d, num_in * sizeof(float)));
     wbCheck(
            hipMalloc((void **)&bin_counts_d, NUM_BINS * sizeof(int)));
    wbCheck(hipMemcpy(in_pos_d, in_pos_h, num_in * sizeof(float),
                         hipMemcpyHostToDevice));
    wbCheck(hipMemset(bin_counts_d, 0, NUM_BINS * sizeof(int)));
    //test scan 
     wbCheck(
          hipMalloc((void **)&bin_ptrs_d, (NUM_BINS + 1) * sizeof(int)));
      wbCheck(hipMemcpy(bin_ptrs_d, bin_ptrs_h,
                         (NUM_BINS + 1) * sizeof(int),
                         hipMemcpyHostToDevice));
    
    
    cpu_preprocess(in_val_h, in_pos_h, in_val_sorted_h, in_pos_sorted_h,
                   grid_size, num_in, bin_counts_h, bin_ptrs_h, in_pos_d, bin_counts_d, bin_ptrs_d);
    wbTime_stop(Generic, "Allocating data for preprocessing");
  }
  
   //test
 // mode =GPU_BINNED_GPU_PREPROCESSING;
  
  // Allocate device variables
  // ----------------------------------------------

 
  
  if (mode != CPU_NORMAL) {

    wbTime_start(GPU, "Allocating data");
    // If preprocessing on the CPU, GPU doesn't need the unsorted arrays
    if (mode != GPU_BINNED_CPU_PREPROCESSING) {
      wbCheck(hipMalloc((void **)&in_val_d, num_in * sizeof(float)));
      wbCheck(hipMalloc((void **)&in_pos_d, num_in * sizeof(float)));
    }

    // All modes need the output array
    wbCheck(hipMalloc((void **)&out_d, grid_size * sizeof(float)));

    // Only binning modes need binning information
    if (mode == GPU_BINNED_CPU_PREPROCESSING ||
        mode == GPU_BINNED_GPU_PREPROCESSING) {

      wbCheck(
          hipMalloc((void **)&in_val_sorted_d, num_in * sizeof(float)));

      wbCheck(
          hipMalloc((void **)&in_pos_sorted_d, num_in * sizeof(float)));

      wbCheck(
          hipMalloc((void **)&bin_ptrs_d, (NUM_BINS + 1) * sizeof(int)));

      if (mode == GPU_BINNED_GPU_PREPROCESSING) {
        // Only used in preprocessing but not the actual computation
        wbCheck(
            hipMalloc((void **)&bin_counts_d, NUM_BINS * sizeof(int)));
      }
    }

    hipDeviceSynchronize();
    wbTime_stop(GPU, "Allocating data");
  }

  // Copy host variables to device
  // ------------------------------------------

  if (mode != CPU_NORMAL) {
    wbTime_start(Copy, "Copying data");
    // If preprocessing on the CPU, GPU doesn't need the unsorted arrays
    if (mode != GPU_BINNED_CPU_PREPROCESSING) {
      wbCheck(hipMemcpy(in_val_d, in_val_h, num_in * sizeof(float),
                         hipMemcpyHostToDevice));

      wbCheck(hipMemcpy(in_pos_d, in_pos_h, num_in * sizeof(float),
                         hipMemcpyHostToDevice));
    }

    // All modes need the output array
    wbCheck(hipMemset(out_d, 0, grid_size * sizeof(float)));

    if (mode == GPU_BINNED_CPU_PREPROCESSING) {

      wbCheck(hipMemcpy(in_val_sorted_d, in_val_sorted_h,
                         num_in * sizeof(float), hipMemcpyHostToDevice));

      wbCheck(hipMemcpy(in_pos_sorted_d, in_pos_sorted_h,
                         num_in * sizeof(float), hipMemcpyHostToDevice));

      wbCheck(hipMemcpy(bin_ptrs_d, bin_ptrs_h,
                         (NUM_BINS + 1) * sizeof(int),
                         hipMemcpyHostToDevice));

    } else if (mode == GPU_BINNED_GPU_PREPROCESSING) {
      // If preprocessing on the GPU, bin counts need to be initialized
      //  and nothing needs to be copied
      wbCheck(hipMemset(bin_counts_d, 0, NUM_BINS * sizeof(int)));
    }

    hipDeviceSynchronize();
    wbTime_stop(Copy, "Copying data");
  }

  
 
  // GPU Preprocessing
  // ------------------------------------------------------

  if (mode == GPU_BINNED_GPU_PREPROCESSING) {

    wbTime_start(Compute, "Preprocessing data on the GPU...");

    gpu_preprocess(in_val_d, in_pos_d, in_val_sorted_d, in_pos_sorted_d,
                   grid_size, num_in, bin_counts_d, bin_ptrs_d);
    wbCheck(hipDeviceSynchronize());
    wbTime_stop(Compute, "Preprocessing data on the GPU...");
  }
// mode = GPU_BINNED_GPU_PREPROCESSING;
  // Launch kernel
  // ----------------------------------------------------------

  wbLog(TRACE, "Launching kernel ");

  if (mode == CPU_NORMAL) {
    wbTime_start(Compute, "Performing CPU_NORMAL computation");
    cpu_normal(in_val_h, in_pos_h, out_h, grid_size, num_in);
    wbTime_stop(Compute, "Performing CPU_NORMAL Scatter computation");
  } else if (mode == GPU_NORMAL) {
    wbTime_start(Compute, "Performing GPU_NORMAL computation");
    gpu_normal(in_val_d, in_pos_d, out_d, grid_size, num_in);
    wbCheck(hipDeviceSynchronize());
    wbTime_stop(Compute, "Performing GPU_NORMAL computation");
  } else if (mode == GPU_CUTOFF) {
    wbTime_start(Compute, "Performing GPU_CUTOFF computation");
    gpu_cutoff(in_val_d, in_pos_d, out_d, grid_size, num_in, cutoff2);
    wbCheck(hipDeviceSynchronize());
    wbTime_stop(Compute, "Performing GPU_CUTOFF computation");
  } else if (mode == GPU_BINNED_CPU_PREPROCESSING ||
             mode == GPU_BINNED_GPU_PREPROCESSING) {
    wbTime_start(Compute, "Performing GPU_BINNED_CPU_PREPROCESSING || "
                          "GPU_BINNED_GPU_PREPROCESSING  computation");
    gpu_cutoff_binned(bin_ptrs_d, in_val_sorted_d, in_pos_sorted_d, out_d,
                      grid_size, cutoff2);
    wbCheck(hipDeviceSynchronize());
    wbTime_stop(Compute, "Performing GPU_BINNED_CPU_PREPROCESSING || "
                         "GPU_BINNED_GPU_PREPROCESSING computation");
  } else {
    wbLog(FATAL, "Invalid mode ", mode);
  }

  // Copy device variables from host
  // ----------------------------------------

  if (mode != CPU_NORMAL) {
    wbCheck(hipMemcpy(out_h, out_d, grid_size * sizeof(float),
                       hipMemcpyDeviceToHost));
    wbCheck(hipDeviceSynchronize());
  }

  // Verify correctness
  // -----------------------------------------------------

  wbSolution(args, out_h, grid_size);

  // Free memory
  // ------------------------------------------------------------

  free(in_val_h);
  free(in_pos_h);
  free(out_h);
  if (mode == GPU_BINNED_CPU_PREPROCESSING) {
    free(bin_counts_h);
    free(bin_ptrs_h);
    free(in_val_sorted_h);
    free(in_pos_sorted_h);
  }
  if (mode != CPU_NORMAL) {
    if (mode != GPU_BINNED_CPU_PREPROCESSING) {
      hipFree(in_val_d);
      hipFree(in_pos_d);
    }
    hipFree(out_d);
    if (mode == GPU_BINNED_CPU_PREPROCESSING ||
        mode == GPU_BINNED_GPU_PREPROCESSING) {
      hipFree(in_val_sorted_d);
      hipFree(in_pos_sorted_d);
      hipFree(bin_ptrs_d);
      if (mode == GPU_BINNED_GPU_PREPROCESSING) {
        hipFree(bin_counts_d);
      }
    }
  }

  return 0;
}

